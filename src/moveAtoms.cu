#include "hip/hip_runtime.h"
//
//  moveAtoms.cu
//  nestedDSMC
//
//  Created by Christopher Watkins on 24/03/2015.
//
//

#include "moveAtoms.cuh"

#include "vectorMath.cuh"

#include "declareDeviceConstants.cuh"
#include "declareDeviceParameters.cuh"

void h_moveParticles(struct hipGraphicsResource **cudaPBOres,
					 double3 *d_vel,
					 double3 *d_acc,
					 double timeValue,
					 int numberOfAtoms)
{
	int blockSize;
	int gridSize;
	
#ifdef CUDA7
	int minGridSize;
	
	hipOccupancyMaxPotentialBlockSize(&minGridSize,
									   &blockSize,
									   (const void *) d_moveParticles,
									   0,
									   numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
#else
	int device;
	hipGetDevice ( &device );
	int numSMs;
	hipDeviceGetAttribute(&numSMs,
						   hipDeviceAttributeMultiprocessorCount,
						   device);
	
	gridSize = 256*numSMs;
	blockSize = NUM_THREADS;
#endif
//	std::cout << "gridsize = " << gridSize << " blocksize = " << blockSize << std::endl;
	
	// Map OpenGL buffer object for writing from CUDA
	double3 *d_pos = mapCUDAVBOd3(cudaPBOres);
	
	d_moveParticles<<<gridSize,blockSize>>>(d_pos,
											d_vel,
											d_acc,
											timeValue,
											numberOfAtoms);
	
	//Unmap buffer object
	unmapCUDAVBO(cudaPBOres);
	
	return;
}

__global__ void d_moveParticles(double3 *pos,
								double3 *vel,
								double3 *acc,
								double dt,
								int numberOfAtoms)
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < numberOfAtoms;
         atom += blockDim.x * gridDim.x)
	{
		double3 l_pos = pos[atom];
		double3 l_vel = vel[atom];
		double3 l_acc = acc[atom];
		
		for (int i=0; i<30000; i++) {
			velocityVerletUpdate(&l_pos,
								 &l_vel,
								 &l_acc,
								 dt);
		}
	
		pos[atom] = l_pos;
		vel[atom] = l_vel;
		acc[atom] = l_acc;
	}

    return;
	
}

__device__ void velocityVerletUpdate(double3 *pos,
									 double3 *vel,
									 double3 *acc,
									 double dt)
{
	vel[0] = updateVel(vel[0],
					   acc[0],
					   0.5*dt);
	pos[0] = updatePos(pos[0],
					   vel[0],
					   dt);
	acc[0] = updateAcc(pos[0]);
	vel[0] = updateVel(vel[0],
					   acc[0],
					   0.5*dt);
	
	return;
}

__device__ void symplecticEulerUpdate(double3 *pos,
									  double3 *vel,
									  double3 *acc,
									  double dt)
{
	acc[0] = updateAcc(pos[0]);
	vel[0] = updateVel(vel[0],
					   acc[0],
					   dt);
	pos[0] = updatePos(pos[0],
					   vel[0],
					   dt);
}

__device__ double3 updateVel(double3 vel,
							 double3 acc,
							 double dt)
{
	return vel + acc * dt;
}

__device__ double3 updatePos(double3 pos,
							 double3 vel,
							 double dt)
{
	return pos + vel * dt;
}

__device__ double3 updateAcc(double3 pos)
{
	
	return -0.5 * d_gs * d_muB * dabsB(pos) / d_mRb;
}